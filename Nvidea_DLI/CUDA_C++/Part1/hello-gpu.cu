
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}


 // `helloGPU` is a kernel that can be launched on the GPU. It must be with a '__global__' keyword and a void return type
 
 __global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloCPU();

  // 'helloGPU' is launched as a kernal on the GPU with 1 block of threads with 1 thread each.

  helloGPU<<<1,1>>>();

  // This synchronizes the completion of the kernal before continuing the CPU thread
  hipDeviceSynchronize();
   
}

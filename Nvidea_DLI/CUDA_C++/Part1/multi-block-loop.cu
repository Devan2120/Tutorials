
#include <hip/hip_runtime.h>
#include <stdio.h>

  /*
   * We are trying to replace the for loop
   * so it can be accelerated.
   * The code prints numbers from 0 to 9 using multiple blocks.
   */

__global__ void loop()
{
  /*
   * This idiomatic expression gives each thread
   * a unique index within the entire grid.
   */

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d\n", i);
}

int main()
{
  /*
   * Additional execution configurations that would
   * work and meet the exercises contraints are:
   *
   * <<<5, 2>>>
   * <<<10, 1>>>
   */
  
  loop<<<2, 5>>>();
  hipDeviceSynchronize();
}
